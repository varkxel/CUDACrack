#include "hip/hip_runtime.h"
#include "Crack.cuh"
#include "Crypt.cuh"

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>

// Constants
#define CharIterations ('Z' - 'A') + 1
#define NumIterations  ('9' - '0') + 1
#define MaxIndex       CharIterations * CharIterations * NumIterations * NumIterations

// GPU Constants
__device__ const long CharIterations_CUDA = CharIterations;
__device__ const long NumIterations_CUDA = NumIterations;
__device__ const long MaxIndex_CUDA = MaxIndex;

__device__ char device_result[5];
__device__ bool result_found = false;

__global__ void CUDACrack(const char* encryptedPassword)
{
	// Get the current index & return if out of bounds
	long index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index >= MaxIndex) return;
	
	// Create the string to encrypt
	char* str = "AA00";
	str[3] += (index) % NumIterations_CUDA;
	str[2] += (index / (NumIterations_CUDA)) % NumIterations_CUDA;
	str[1] += (index / (NumIterations_CUDA * NumIterations_CUDA)) % CharIterations_CUDA;
	str[0] += (index / (NumIterations_CUDA * NumIterations_CUDA * CharIterations_CUDA)) % CharIterations_CUDA;
	
	// Get the encrypted string
	char encrypted[11];
	CUDACrypt(str, encrypted);
	
	// strcomp()
	bool equal = true;
	for(int i = 0; i < 11; i++) equal &= encryptedPassword[i] == encrypted[i];
	
	if(equal)
	{
		// Copy string to result
		for(int i = 0; i < 4; i++) device_result[i] = str[i];
		device_result[4] = '\0';
		result_found = true;
	}
}

int main(int argc, char** argv)
{
	bool encrypted_allocated = false;
	char* encrypted = NULL;
	if(argc < 2)
	{
		printf("Enter encrypted password to crack: ");
		encrypted = (char*) malloc(128 * sizeof(char));
		encrypted[127] = '\0';
		scanf("%127s", encrypted);
		encrypted_allocated = true;
	}
	else encrypted = argv[1];
	
	const size_t encrypted_length = strlen(encrypted);
	const size_t encrypted_allocSize = encrypted_length + 1;
	
	char* encrypted_device;
	hipMalloc((void**) &encrypted_device, sizeof(char) * encrypted_allocSize);
	hipMemcpy(encrypted_device, encrypted, sizeof(char) * encrypted_allocSize, hipMemcpyHostToDevice);
	
	// Schedule CUDA job
	CUDACrack<<<CharIterations * CharIterations, NumIterations * NumIterations>>>(encrypted);
	
	// Cleanup
	hipFree(encrypted_device);
	if(encrypted_allocated) free(encrypted);
	
	// Copy result back
	bool resultFound;
	hipMemcpyFromSymbol(&resultFound, HIP_SYMBOL("result_found"), sizeof(bool), 0, hipMemcpyDeviceToHost);
	
	if(resultFound)
	{
		char result[5];
		hipMemcpyFromSymbol(result, HIP_SYMBOL("device_result"), sizeof(char) * 5, 0, hipMemcpyDeviceToHost);
		printf("%s\n", result);
	}
	else printf("Could not crack password.\n");
}
